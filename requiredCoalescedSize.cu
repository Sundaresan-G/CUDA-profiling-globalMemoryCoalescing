/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


// In this program, consecutive group of threads access either 32-byte, 64-byte or 128-byte. 
// But subsequent group of threads shall access farther memory locations.
// For 32 byte example, threads 0-7 access 4 byte each at memory location 0, but threads 8-16 access 4 byte each at memory location 64 and so on.
// Use only float datatype

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

template <typename T>
__global__ void coalescedAccessCheck(T* a, int s)
{
  // s is either 32, 64 or 128
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  s /= 4; // either 8 threads access 32 byte or 16 threads access 64 bytes
  int j = i%s + (i/s)*s*2;
  a[j] = a[j] + 1;
}

template <typename T>
void runTest(int deviceId, int nMB)
{
  int blockSize = 512;
  float ms;

  T *d_a;
  hipEvent_t startEvent, stopEvent;
    
  int n = nMB*1024*1024/sizeof(T);

  // NB:  d_a(33*nMB) for stride case
  checkCuda( hipMalloc(&d_a, n * 33 * sizeof(T)) );

  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );

  printf("CoalescedBytes, Bandwidth (GB/s):\n");
  
  coalescedAccessCheck<<<n/blockSize, blockSize>>>(d_a, 32); // warm up

  // Here i specifies the number of bytes of contiguous access
  for (int i = 4; i <= 128; i = i << 1) {
    checkCuda( hipMemset(d_a, 0, n * sizeof(T)) );

    checkCuda( hipEventRecord(startEvent,0) );
    coalescedAccessCheck<<<n/blockSize, blockSize>>>(d_a, i);
    checkCuda( hipEventRecord(stopEvent,0) );
    checkCuda( hipEventSynchronize(stopEvent) );

    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    printf("%13d, %f\n", i, 2*nMB/ms);
  }

  printf("\n");

  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  hipFree(d_a);
}

int main(int argc, char **argv)
{
  int nMB = 4 * 16;
  int deviceId = 0;
  bool bFp64 = false;

  for (int i = 1; i < argc; i++) {    
    if (!strncmp(argv[i], "dev=", 4))
      deviceId = atoi((char*)(&argv[i][4]));
    else if (!strcmp(argv[i], "fp64"))
      bFp64 = true;
  }
  
  hipDeviceProp_t prop;
  
  checkCuda( hipSetDevice(deviceId) );

  checkCuda( hipGetDeviceProperties(&prop, deviceId) );
  printf("Device: %s\n", prop.name);
  printf("Transfer size (MB): %d\n", nMB);
  
  printf("%s Precision\n", bFp64 ? "Double" : "Single");
  
  if (bFp64) runTest<double>(deviceId, nMB);
  else       runTest<float>(deviceId, nMB);
}